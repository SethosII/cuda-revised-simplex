#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>

#include "cudaCheck.cuh"
#include "lpProblem.cuh"
#include "print.cuh"
#include "simplex.cuh"

int32_t main(int32_t argc, char *argv[]) {
	hipsolverSpHandle_t cusolverHandle;
	cuSolverCheck(hipsolverSpCreate(&cusolverHandle));
	hipsparseHandle_t cusparseHandle;
	cuSparseCheck(hipsparseCreate(&cusparseHandle));

	hipsparseMatDescr_t matrixDescriptor;
	cuSparseCheck(hipsparseCreateMatDescr(&matrixDescriptor));
	cuSparseCheck(
			hipsparseSetMatType(matrixDescriptor, HIPSPARSE_MATRIX_TYPE_GENERAL));
	cuSparseCheck(
			hipsparseSetMatIndexBase(matrixDescriptor,
					HIPSPARSE_INDEX_BASE_ZERO));

	LPProblem *lpProblem = (LPProblem *) malloc(sizeof(LPProblem));
	readMPS(argv[1], lpProblem);
	LPProblem *lpProblemMod = (LPProblem *) malloc(sizeof(LPProblem));
	convertToStandardform(lpProblem, lpProblemMod);
	findBFS(lpProblemMod, cusolverHandle, cusparseHandle, matrixDescriptor);
	LPProblem *lpProblemCopy = (LPProblem *) malloc(sizeof(LPProblem));
	copyLPProblem(lpProblemMod, lpProblemCopy);
	deleteLPProblem(lpProblem);
	deleteLPProblem(lpProblemMod);
	deleteLPProblem(lpProblemCopy);

	cudaCheck(hipDeviceReset());

	return EXIT_SUCCESS;
}
