/**
 * Implements LP problem
 */

#include <glpk.h>
#include <stdio.h>

#include "cudaCheck.cuh"
#include "lpProblem.cuh"
#include "print.cuh"

void readMPS(char *mpsFile, LPProblem *lpProblem) {
	glp_prob *lp = glp_create_prob();
	glp_read_mps(lp, GLP_MPS_FILE, NULL, mpsFile);
	lpProblem->rows = glp_get_num_rows(lp);
	lpProblem->columns = glp_get_num_cols(lp);
	lpProblem->nnz = glp_get_num_nz(lp);
	lpProblem->isBasisAllocated = false;


	cudaCheck(
			hipMallocManaged(&lpProblem->A, lpProblem->rows * lpProblem->columns * sizeof(double)));
	cudaCheck(hipMallocManaged(&lpProblem->b, lpProblem->rows * sizeof(double)));
	cudaCheck(hipMallocManaged(&lpProblem->c, lpProblem->columns * sizeof(double)));
	cudaCheck(
			hipMallocManaged(&lpProblem->lowerBound,
					lpProblem->columns * sizeof(double)));
	cudaCheck(
			hipMallocManaged(&lpProblem->upperBound,
					lpProblem->columns * sizeof(double)));

	for (int32_t i = 0; i < lpProblem->rows * lpProblem->columns; i++) {
		lpProblem->A[i] = 0.;
	}
	int32_t *indices = (int32_t *) malloc(lpProblem->columns * sizeof(int32_t));
	double *values = (double *) malloc(lpProblem->columns * sizeof(double));
	// for glpk i + 1 (indices one-based)
	for (int32_t i = 0; i < lpProblem->rows; i++) {
		if (glp_get_row_type(lp, i + 1) == GLP_FR) {
			// ignore cost row
			printf("shouldn't be here!!\n");
			continue;
		}
		int32_t numberValues = glp_get_mat_row(lp, i + 1, indices, values);
		for (uint32_t j = 0; j < numberValues; j++) {
			lpProblem->A[i * lpProblem->columns + indices[j + 1] - 1] = values[j + 1];
		}
	}
	free(indices);
	free(values);

	for (int32_t i = 0; i < lpProblem->rows; i++) {
		// constraints are expected to be in form A*x=b
		if (glp_get_row_type(lp, i + 1) == GLP_FX) {
			lpProblem->b[i] = glp_get_row_lb(lp, i + 1);
		} else {
			printf("Can only handle constraints in form A*x=b!");
			exit(EXIT_FAILURE);
		}
	}

	for (uint32_t i = 0; i < lpProblem->columns; i++) {
		lpProblem->c[i] = glp_get_obj_coef(lp, i + 1);
	}

	for (int32_t i = 0; i < lpProblem->columns; i++) {
		lpProblem->lowerBound[i] = glp_get_col_lb(lp, i + 1);
		lpProblem->upperBound[i] = glp_get_col_ub(lp, i + 1);
	}

	glp_delete_prob(lp);
	glp_free_env();
}

void convertToStandardform(LPProblem *source, LPProblem *converted) {
	converted->isBasisAllocated = source->isBasisAllocated;
	converted->rows= source->rows + source->columns;
	converted->columns = source->columns * 2;
	converted->nnz = source->nnz + source->columns * 2;
	cudaCheck(hipMallocManaged(&converted->A, converted->rows * converted->columns * sizeof(double)));
	for (int32_t i = 0; i < converted->rows; i++) {
		for (int32_t j = 0; j < converted->columns; j++) {
			if (i < source->rows && j < source->columns) {
				converted->A[i * converted->columns + j] = source->A[i * source->columns + j];
			} else if(i < source->rows) {
				converted->A[i * converted->columns + j] = 0;
			} else if(j < source->columns) {
				if (i - source->rows == j) {
					converted->A[i * converted->columns + j] = 1;
				} else {
					converted->A[i * converted->columns + j] = 0;
				}
			} else {
				if (i - source->rows == j - source->columns) {
					converted->A[i * converted->columns + j] = 1;
				} else {
					converted->A[i * converted->columns + j] = 0;
				}
			}
		}
	}
	cudaCheck(hipMallocManaged(&converted->b, converted->rows * sizeof(double)));
	for (int32_t i = 0; i < converted->rows; i++) {
		if (i < source->rows) {
			converted->b[i] = source->b[i];
			for (int32_t j = 0; j < source->columns; j++) {
				converted->b[i] -= source->A[i * source->columns + j] * source->lowerBound[j];
			}
		} else {
			converted->b[i] = source->upperBound[i - source->rows] - source->lowerBound[i - source->rows];
		}
		if (converted->b[i] < 0) {
			converted->b[i] = -converted->b[i];
			for (int j = 0; j < converted->columns; j++) {
				converted->A[i * converted->columns + j] = -converted->A[i * converted->columns + j];
			}
		}
	}
	cudaCheck(hipMallocManaged(&converted->c, converted->columns * sizeof(double)));
	for (int32_t i = 0; i < converted->columns; i++) {
		if (i < source->columns) {
			converted->c[i] = source->c[i];
		} else {
			converted->c[i] = 0;
		}
	}
}

void copyLPProblem(LPProblem *source, LPProblem *destination) {
	if (source->isBasisAllocated) {
		initializeLPProblem(destination, source->rows, source->columns, source->nnz);
		destination->isBasisAllocated = true;
		destination->rows = source->rows;
		destination->columns = source->columns;
		destination->nnz = source->nnz;
		cudaCheck(hipMemcpy(destination->A, source->A, destination->rows * destination->columns * sizeof(double), hipMemcpyDeviceToDevice));
		cudaCheck(hipMemcpy(destination->b, source->b, destination->rows * sizeof(double), hipMemcpyDeviceToDevice));
		cudaCheck(hipMemcpy(destination->c, source->c, destination->columns * sizeof(double), hipMemcpyDeviceToDevice));

		destination->nnzAB = source->nnzAB;
		cudaCheck(hipMemcpy(destination->AB, source->AB, destination->rows * destination->rows * sizeof(double), hipMemcpyDeviceToDevice));
		cudaCheck(hipMemcpy(destination->ABRowPointer, source->ABRowPointer, (destination->rows + 1) * sizeof(int32_t), hipMemcpyDeviceToDevice));
		cudaCheck(hipMemcpy(destination->ABColumnIndices, source->ABColumnIndices, destination->nnz * sizeof(int32_t), hipMemcpyDeviceToDevice));
		cudaCheck(hipMemcpy(destination->ABValues, source->ABValues, destination->nnz * sizeof(double), hipMemcpyDeviceToDevice));

		cudaCheck(hipMemcpy(destination->ABTRowPointer, source->ABTRowPointer, (destination->rows + 1) * sizeof(int32_t), hipMemcpyDeviceToDevice));
		cudaCheck(hipMemcpy(destination->ABTColumnIndices, source->ABTColumnIndices, destination->nnz * sizeof(int32_t), hipMemcpyDeviceToDevice));
		cudaCheck(hipMemcpy(destination->ABTValues, source->ABTValues, destination->nnz * sizeof(double), hipMemcpyDeviceToDevice));

		destination->nnzANB = source->nnzANB;
		cudaCheck(hipMemcpy(destination->ANB, source->ANB, destination->rows * (destination->columns - destination->rows) * sizeof(double), hipMemcpyDeviceToDevice));
		cudaCheck(hipMemcpy(destination->ANBRowPointer, source->ANBRowPointer, (destination->rows + 1) * sizeof(int32_t), hipMemcpyDeviceToDevice));
		cudaCheck(hipMemcpy(destination->ANBColumnIndices, source->ANBColumnIndices, destination->nnz * sizeof(int32_t), hipMemcpyDeviceToDevice));
		cudaCheck(hipMemcpy(destination->ANBValues, source->ANBValues, destination->nnz * sizeof(double), hipMemcpyDeviceToDevice));

		cudaCheck(hipMemcpy(destination->cB, source->cB, destination->rows * sizeof(double), hipMemcpyDeviceToDevice));
		cudaCheck(hipMemcpy(destination->cBIndex, source->cBIndex, destination->rows * sizeof(int32_t), hipMemcpyDeviceToDevice));
		cudaCheck(hipMemcpy(destination->cNB, source->cNB, (destination->columns - destination->rows) * sizeof(double), hipMemcpyDeviceToDevice));
		cudaCheck(hipMemcpy(destination->cNBIndex, source->cNBIndex, (destination->columns - destination->rows) * sizeof(int32_t), hipMemcpyDeviceToDevice));


		cudaCheck(hipMemcpy(destination->xB, source->xB, destination->rows * sizeof(double), hipMemcpyDeviceToDevice));
		cudaCheck(hipMemcpy(destination->xIndex, source->xIndex, destination->columns * sizeof(int32_t), hipMemcpyDeviceToDevice));
		cudaCheck(hipMemcpy(destination->lowerBound, source->lowerBound, destination->columns * sizeof(double), hipMemcpyDeviceToDevice));
		cudaCheck(hipMemcpy(destination->upperBound, source->upperBound, destination->columns * sizeof(double), hipMemcpyDeviceToDevice));
	} else {
		destination->isBasisAllocated = false;
		destination->rows = source->rows;
		destination->columns = source->columns;
		destination->nnz = source->nnz;
		cudaCheck(hipMallocManaged(&destination->A, destination->rows * destination->columns * sizeof(double)));
		cudaCheck(hipMemcpy(destination->A, source->A, destination->rows * destination->columns * sizeof(double), hipMemcpyDeviceToDevice));
		cudaCheck(hipMallocManaged(&destination->b, destination->rows * sizeof(double)));
		cudaCheck(hipMemcpy(destination->b, source->b, destination->rows * sizeof(double), hipMemcpyDeviceToDevice));
		cudaCheck(hipMallocManaged(&destination->c, destination->columns * sizeof(double)));
		cudaCheck(hipMemcpy(destination->c, source->c, destination->columns * sizeof(double), hipMemcpyDeviceToDevice));
	}
}

void deleteLPProblem(LPProblem *lpProblem) {
	hipFree(lpProblem->A);
	hipFree(lpProblem->b);
	hipFree(lpProblem->c);
	hipFree(lpProblem->lowerBound);
	hipFree(lpProblem->upperBound);
	if (lpProblem->isBasisAllocated) {
		hipFree(lpProblem->AB);
		hipFree(lpProblem->ABRowPointer);
		hipFree(lpProblem->ABColumnIndices);
		hipFree(lpProblem->ABValues);
		hipFree(lpProblem->ABTRowPointer);
		hipFree(lpProblem->ABTColumnIndices);
		hipFree(lpProblem->ABTValues);
		hipFree(lpProblem->ANB);
		hipFree(lpProblem->ANBRowPointer);
		hipFree(lpProblem->ANBColumnIndices);
		hipFree(lpProblem->ANBValues);
		hipFree(lpProblem->cB);
		hipFree(lpProblem->cBIndex);
		hipFree(lpProblem->cNB);
		hipFree(lpProblem->cNBIndex);
		hipFree(lpProblem->xB);
		hipFree(lpProblem->xIndex);
		hipFree(lpProblem->s);
		hipFree(lpProblem->g);
		hipFree(lpProblem->gTemp);
		hipFree(lpProblem->ANBColumn);
		hipFree(lpProblem->nnzPerRow);
		hipFree(lpProblem->row);
		hipFree(lpProblem->column);
	}
	free(lpProblem);
}

void initializeLPProblem(LPProblem *lpProblem, int32_t rows, int32_t columns,
		int32_t nnz) {
	lpProblem->isBasisAllocated = true;
	lpProblem->rows = rows;
	lpProblem->columns = columns;
	lpProblem->nnz = nnz;
	cudaCheck(
			hipMallocManaged(&lpProblem->A, rows * columns * sizeof(double)));
	cudaCheck(hipMallocManaged(&lpProblem->b, rows * sizeof(double)));
	cudaCheck(hipMallocManaged(&lpProblem->c, columns * sizeof(double)));
	cudaCheck(
			hipMallocManaged(&lpProblem->lowerBound,
					columns * sizeof(double)));
	cudaCheck(
			hipMallocManaged(&lpProblem->upperBound,
					columns * sizeof(double)));

	cudaCheck(hipMallocManaged(&lpProblem->AB, rows * rows * sizeof(double)));
	cudaCheck(
			hipMallocManaged(&lpProblem->ABRowPointer,
					(rows + 1) * sizeof(int32_t)));
	cudaCheck(
			hipMallocManaged(&lpProblem->ABColumnIndices,
					nnz * sizeof(int32_t)));
	cudaCheck(hipMallocManaged(&lpProblem->ABValues, nnz * sizeof(double)));

	cudaCheck(
			hipMallocManaged(&lpProblem->ABTRowPointer,
					(rows + 1) * sizeof(int32_t)));
	cudaCheck(
			hipMallocManaged(&lpProblem->ABTColumnIndices,
					nnz * sizeof(int32_t)));
	cudaCheck(hipMallocManaged(&lpProblem->ABTValues, nnz * sizeof(double)));

	cudaCheck(
			hipMallocManaged(&lpProblem->ANB,
					rows * (columns - rows) * sizeof(double)));
	cudaCheck(
			hipMallocManaged(&lpProblem->ANBRowPointer,
					(rows + 1) * sizeof(int32_t)));
	cudaCheck(
			hipMallocManaged(&lpProblem->ANBColumnIndices,
					nnz * sizeof(int32_t)));
	cudaCheck(hipMallocManaged(&lpProblem->ANBValues, nnz * sizeof(double)));

	cudaCheck(hipMallocManaged(&lpProblem->cB, rows * sizeof(double)));
	cudaCheck(hipMallocManaged(&lpProblem->cBIndex, rows * sizeof(int32_t)));
	cudaCheck(
			hipMallocManaged(&lpProblem->cNB,
					(columns - rows) * sizeof(double)));
	cudaCheck(
			hipMallocManaged(&lpProblem->cNBIndex,
					(columns - rows) * sizeof(int32_t)));

	cudaCheck(hipMallocManaged(&lpProblem->xB, rows * sizeof(double)));
	cudaCheck(hipMallocManaged(&lpProblem->xIndex, columns * sizeof(int32_t)));
	cudaCheck(hipMallocManaged(&lpProblem->s, rows * sizeof(double)));
	cudaCheck(
			hipMallocManaged(&lpProblem->g,
					(columns - rows) * sizeof(double)));
	cudaCheck(
			hipMallocManaged(&lpProblem->gTemp,
					(columns - rows) * sizeof(double)));
	cudaCheck(hipMallocManaged(&lpProblem->ANBColumn, rows * sizeof(double)));
	cudaCheck(hipMallocManaged(&lpProblem->nnzPerRow, rows * sizeof(int32_t)));

	cudaCheck(hipMallocManaged(&lpProblem->row, sizeof(int32_t)));
	cudaCheck(hipMallocManaged(&lpProblem->column, sizeof(int32_t)));
}
